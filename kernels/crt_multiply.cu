#include "hip/hip_runtime.h"
#include "kernel_interface.h"
#include "helpers.h"
#include <stdio.h>
#include <stdint.h>
#include <iostream>
__global__ void multiplyKernel(
    uint32_t* C,
    const uint32_t* A,
    const uint32_t* B,
    size_t sizeA,
    size_t sizeB,
    uint64_t* moduli,
    size_t numModuli,
    uint64_t* W,
    uint32_t* accum,
    uint32_t* temp
) {
    const size_t sizeC = sizeA + sizeB;
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ uint64_t residues[256];
    
    if (idx < numModuli) {
        uint64_t modulus = moduli[idx];

        uint64_t r_A = 0;
        uint64_t r_B = 0;

        for (int i = 0; i < sizeA; i++) {
            r_A = (r_A << 32) % modulus;
            r_A = (uint64_t)(((unsigned __int128)r_A + A[i]) % modulus);
        }
        for (int i = 0; i < sizeB; i++) {
            r_B = (r_B << 32) % modulus;
            r_B = (uint64_t)(((unsigned __int128)r_B + B[i]) % modulus);
        }
        
        uint64_t product = (uint64_t)(((unsigned __int128)r_A * r_B) % modulus);
        residues[threadIdx.x] = product;
    }
    __syncthreads();

    /*
        Until I figure out how to parallelize this it will just be done on one thread.
        We are using Garner's algorithm here, which means we will convert the product
        from modular form to mixed radix form, then recombine.
    */
    __shared__ uint64_t x[256];
    
    if (idx == 0) {
        for (int i = 0; i < numModuli; i++) {
            x[i] = residues[i];
            for (int j = 0; j < i; j++) {
                uint64_t inverse = W[j * numModuli + i];
                x[i] = (uint64_t)(((unsigned __int128)(x[i] - x[j] + moduli[i]) * inverse) % moduli[i]);
            }
        }
        
        // Initialize output array
        for (size_t i = 0; i < sizeC; i++) {
            C[i] = 0;
        }
        
        uint64_t accum_len = 1;

        for (int i = 0; i < sizeC; i++) {
            accum[i] = 0;
        }

        uint64_t temp_len = 0;
        for (int i = numModuli - 1; i >= 0; i--) {
            multi_precision_multiply(accum, accum_len, moduli[i], temp, &temp_len);
            multi_precision_add(temp, temp_len, x[i], accum, &accum_len);
        }

        for (int i = 0; i < sizeC; i++) {
        if (i < accum_len) {
            C[i] = accum[i];
            }
            else {
                C[i] = 0;
            }
        }
    }
}

extern "C" hipError_t multiply(
    uint32_t* C,
    const uint32_t* A,
    const uint32_t* B,
    size_t sizeA,
    size_t sizeB
) {
    /*  
        We pick a set of moduli that are all approximately 2^64, and we want to ensure
        that their product is greater than A * B. The product has at most sizeA + sizeB chunks
        = 32 * (sizeA + sizeB) bits, and each modulus contributes 64 bits to the product of the moduli.
        Therefore, we approximately need ceil((sizeA + sizeB) / 2) moduli.
    */
    uint64_t numModuli = (sizeA + sizeB + 1) / 2;
    uint64_t* moduli = new uint64_t[numModuli];

    uint64_t currentCandidate = UINT64_MAX_PRIME;
    uint64_t count = 0;
    while (count < numModuli) {
        if (isPrime(currentCandidate)) {
            moduli[count] = currentCandidate;
            count++;
        }
        currentCandidate--;
    }

    // Compute the inverse matrix W where W[i][j] is the modular inverse of m_i mod m_j
    uint64_t* W = new uint64_t[numModuli * numModuli];
    for (uint64_t i = 0; i < numModuli; i++) {
        for (uint64_t j = 0; j < numModuli; j++) {
            if (i == j) {
                W[i * numModuli + j] = 1;
            } else {
                W[i * numModuli + j] = modInverse(moduli[i] % moduli[j], moduli[j]);
            }
        }
    }
    uint64_t* d_W;
    hipMalloc((void**)&d_W, numModuli * numModuli * sizeof(uint64_t));
    hipMemcpy(d_W, W, numModuli * numModuli * sizeof(uint64_t), hipMemcpyHostToDevice);
    delete[] W;

    uint64_t* d_Moduli;
    hipMalloc((void**)&d_Moduli, numModuli * sizeof(uint64_t));
    hipMemcpy(d_Moduli, moduli, numModuli * sizeof(uint64_t), hipMemcpyHostToDevice);
    delete[] moduli;

    size_t sizeC = sizeA + sizeB;
    uint32_t* d_accum;
    uint32_t* d_temp;
    hipMalloc((void**)&d_accum, sizeC * sizeof(uint32_t));
    hipMalloc((void**)&d_temp, sizeC * sizeof(uint32_t));

    int threadsPerBlock = 256;
    
    size_t totalWork = sizeA * sizeB;
    // 65535 is the maximum number of blocks that can be used in a CUDA kernel
    int numBlocks = min((totalWork + threadsPerBlock - 1) / threadsPerBlock, (size_t)65535);

    multiplyKernel<<<numBlocks, threadsPerBlock>>>(C, A, B, sizeA, sizeB, d_Moduli, numModuli, d_W, d_accum, d_temp);
    
    hipFree(d_Moduli);
    hipFree(d_W);
    hipFree(d_accum);
    hipFree(d_temp);
    return hipGetLastError();
}

extern "C" const char* getKernelName() {
    return "Chinese Remainder Theorem Multiplication";
}

extern "C" const char* getKernelDescription() {
    return "Multiplies two numbers A and B by using the Chinese Remainder Theorem. We compute a set of relatively prime moduli, reduce A and B to their modular representations, multiply the results, then recombine.";
} 