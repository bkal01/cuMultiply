#include "hip/hip_runtime.h"
#include "kernel_interface.h"
#include "host_helpers.h"
#include "device_helpers.h"
#include <stdio.h>
#include <stdint.h>
#include <iostream>
__global__ void multiplyKernel(
    uint32_t* C,
    const uint32_t* A,
    const uint32_t* B,
    size_t sizeA,
    size_t sizeB,
    uint64_t* moduli,
    size_t numModuli,
    uint64_t* W,
    uint32_t* accum,
    uint32_t* temp
) {
    const size_t sizeC = sizeA + sizeB;
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ uint64_t residues[256];
    
    if (idx < numModuli) {
        uint64_t modulus = moduli[idx];

        uint64_t r_A = 0;
        uint64_t r_B = 0;

        for (int i = sizeA; i > 0; i--) {
            r_A = (((unsigned __int128)r_A << 32) % modulus);
            r_A = (r_A + A[i - 1]) % modulus;
        }

        for (int i = sizeB; i > 0; i--) {
            r_B = (((unsigned __int128)r_B << 32) % modulus);
            r_B = (r_B + B[i - 1]) % modulus;
        }
        
        uint64_t product = (uint64_t)(((unsigned __int128)r_A * r_B) % modulus);
        residues[threadIdx.x] = product;
    }
    __syncthreads();

    /*
        Until I figure out how to parallelize this it will just be done on one thread.
        We are using Garner's algorithm here, which means we will convert the product
        from modular form to mixed radix form, then recombine.
    */
    __shared__ uint64_t x[256];
    
    if (idx == 0) {
        for (int i = 0; i < numModuli; i++) {
            x[i] = residues[i];
            for (int j = 0; j < i; j++) {
                uint64_t inverse = W[j * numModuli + i];
                x[i] = (uint64_t)(((unsigned __int128)(x[i] + moduli[i] - x[j]) * inverse) % moduli[i]);
            }
        }
        
        // Initialize accumulator (assumed to be little‑endian; starting at 0)
        for (size_t i = 0; i < sizeC; i++) {
            C[i] = 0;
            accum[i] = 0;
        }
        uint64_t accum_len = 1;  // accum currently has one 32-bit word (0)

        uint64_t temp_len = 0;
        // Garner reconstruction: accum = accum * moduli[i] + x[i]
        for (int i = (int)numModuli - 1; i >= 0; i--) {
            // Use the new helper; note that the result length is accum_len + 2 (max)
            multi_precision_multiply(accum, accum_len, moduli[i], temp, &temp_len);
            while (temp_len > 1 && temp[temp_len - 1] == 0)
                temp_len--;
            // Add x[i] (converted to multi‑precision form) to temp.
            // (Assume multi_precision_add handles a 64-bit add and updates accum_len.)
            multi_precision_add(temp, temp_len, x[i], accum, &accum_len);
        }

        for (uint32_t i = 0; i < sizeC; i++) {
            C[i] = (i < accum_len) ? accum[i] : 0;
        }
    }
}


extern "C" hipError_t multiply(
    uint32_t* C,
    const uint32_t* A,
    const uint32_t* B,
    size_t sizeA,
    size_t sizeB
) {
    /*  
        We pick a set of moduli that are all approximately 2^64, and we want to ensure
        that their product is greater than A * B. The product has at most sizeA + sizeB chunks
        = 32 * (sizeA + sizeB) bits, and each modulus contributes 64 bits to the product of the moduli.
        Therefore, we approximately need ceil((sizeA + sizeB) / 2) moduli.
    */
    uint64_t numModuli = (sizeA + sizeB + 1) / 2;
    uint64_t* moduli = new uint64_t[numModuli];

    uint64_t currentCandidate = UINT64_MAX_PRIME;
    uint64_t count = 0;
    while (count < numModuli) {
        if (isPrime(currentCandidate)) {
            moduli[count] = currentCandidate;
            count++;
        }
        currentCandidate--;
    }

    // Compute the inverse matrix W where W[i][j] is the modular inverse of m_i mod m_j
    uint64_t* W = new uint64_t[numModuli * numModuli];
    for (uint64_t i = 0; i < numModuli; i++) {
        for (uint64_t j = 0; j < numModuli; j++) {
            if (i == j) {
                W[j * numModuli + i] = 1;
            } else if (j < i) {
                W[j * numModuli + i] = modInverse(moduli[j] % moduli[i], moduli[i]);
            }
        }
    }
    uint64_t* d_W;
    hipMalloc((void**)&d_W, numModuli * numModuli * sizeof(uint64_t));
    hipMemcpy(d_W, W, numModuli * numModuli * sizeof(uint64_t), hipMemcpyHostToDevice);
    delete[] W;

    uint64_t* d_Moduli;
    hipMalloc((void**)&d_Moduli, numModuli * sizeof(uint64_t));
    hipMemcpy(d_Moduli, moduli, numModuli * sizeof(uint64_t), hipMemcpyHostToDevice);
    delete[] moduli;

    size_t sizeC = sizeA + sizeB;
    uint32_t* d_accum;
    uint32_t* d_temp;
    hipMalloc((void**)&d_accum, sizeC * sizeof(uint32_t));
    hipMalloc((void**)&d_temp, sizeC * sizeof(uint32_t));

    int threadsPerBlock = 256;
    
    // 65535 is the maximum number of blocks that can be used in a CUDA kernel
    int numBlocks = min((numModuli + threadsPerBlock - 1) / threadsPerBlock, (size_t)65535);

    multiplyKernel<<<numBlocks, threadsPerBlock>>>(C, A, B, sizeA, sizeB, d_Moduli, numModuli, d_W, d_accum, d_temp);
    
    hipFree(d_Moduli);
    hipFree(d_W);
    hipFree(d_accum);
    hipFree(d_temp);
    return hipGetLastError();
}

extern "C" const char* getKernelName() {
    return "Chinese Remainder Theorem Multiplication";
}

extern "C" const char* getKernelDescription() {
    return "Multiplies two numbers A and B by using the Chinese Remainder Theorem. We compute a set of relatively prime moduli, reduce A and B to their modular representations, multiply the results, then recombine.";
} 