#include "hip/hip_runtime.h"
#include "device_helpers.h"

extern "C" {

__global__ void test_multi_precision_multiply_kernel(
    const uint32_t *input,
    size_t length,
    uint64_t multiplier,
    uint32_t *result,
    uint64_t *result_len
) {
    if (threadIdx.x == 0) {
        multi_precision_multiply(input, length, multiplier, result, result_len);
    }
}

__global__ void test_multi_precision_add_kernel(
    const uint32_t *a,
    int a_len,
    uint64_t addend,
    uint32_t *result,
    uint64_t *result_len
) {
    if (threadIdx.x == 0) {
        multi_precision_add(a, a_len, addend, result, result_len);
    }
}

__global__ void test_multi_precision_add_arrays_kernel(
    uint32_t *a,
    uint64_t a_len,
    const uint32_t *b,
    uint64_t b_len,
    uint64_t *result_len
) {
    if (threadIdx.x == 0) {
        multi_precision_add_arrays(a, a_len, b, b_len, result_len);
    }
}

hipError_t launch_multi_precision_multiply_test(
    const uint32_t *input,
    size_t length,
    uint64_t multiplier,
    uint32_t *result,
    uint64_t *result_len
) {
    test_multi_precision_multiply_kernel<<<1, 32>>>(input, length, multiplier, result, result_len);
    return hipGetLastError();
}

hipError_t launch_multi_precision_add_test(
    const uint32_t *a,
    int a_len,
    uint64_t addend,
    uint32_t *result,
    uint64_t *result_len
) {
    test_multi_precision_add_kernel<<<1, 32>>>(a, a_len, addend, result, result_len);
    return hipGetLastError();
}

hipError_t launch_multi_precision_add_arrays_test(
    uint32_t *a,
    uint64_t a_len,
    const uint32_t *b,
    uint64_t b_len,
    uint64_t *result_len
) {
    test_multi_precision_add_arrays_kernel<<<1, 32>>>(a, a_len, b, b_len, result_len);
    return hipGetLastError();
}

}