#include "hip/hip_runtime.h"
#include "kernel_interface.h"
#include <stdio.h>


__global__ void multiply_kernel(uint32_t* C, const uint32_t* A, const uint32_t* B, size_t sizeA, size_t sizeB) {
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        C[0] = -1;
    }
}

extern "C" hipError_t multiply(
    uint32_t* C,
    uint64_t* bigC,
    const uint32_t* A,
    const uint32_t* B,
    size_t sizeA,
    size_t sizeB,
    hipStream_t stream
) {
    int numBlocks = 1;
    int threadsPerBlock = 256;
    
    multiply_kernel<<<numBlocks, threadsPerBlock>>>(C, A, B, sizeA, sizeB);
    
    return hipGetLastError();
}

extern "C" const char* getKernelName() {
    return "Incorrect Multiplication";
}

extern "C" const char* getKernelDescription() {
    return "Incorrectly sets the result to be -1.";
} 