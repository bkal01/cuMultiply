#include "hip/hip_runtime.h"
#include "kernel_interface.h"
#include <stdio.h>


__global__ void multiply_kernel(int* C, const int* A, const int* B, size_t sizeA, size_t sizeB) {
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        C[0] = -1;
    }
}

extern "C" hipError_t multiply(
    int* C, 
    const int* A, 
    const int* B, 
    size_t sizeA,
    size_t sizeB,
    hipStream_t stream
) {
    int numBlocks = 1;
    int threadsPerBlock = 256;
    
    multiply_kernel<<<numBlocks, threadsPerBlock>>>(C, A, B, sizeA, sizeB);
    
    return hipGetLastError();
}

extern "C" const char* getKernelName() {
    return "Incorrect Multiplication";
}

extern "C" const char* getKernelDescription() {
    return "Incorrectly sets the result to be -1.";
} 